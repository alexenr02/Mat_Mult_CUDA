
#include "main.cuh"
#include "file_handler.cuh"
#include "user_input_handler.cuh"
#include "math_operation.cuh"

/*
* Function that executes the reading process of the two txt files that contains the arrays of array
* and saves them in dynamic memory arrays of arrays.
*/
processStatus_t loadTxts(matrix_t matrix_data[])
{

    _Post_ _Notnull_ FILE* filePointer; /* declare the file pointer */
    FILE* filePointer_aux = NULL;
    uint8_t readStatus = 0;


    const char* stringArray[NUM_FILES] =        //Setting an array with the names of the two txt files
    {
        MATRIXA_FILENAME,
        MATRIXB_FILENAME
    };


    //loop to read two files that contains many double (8 byte) variables
    for (uint8_t i = 0; i < NUM_FILES; i++)
    {
        printf("\n---Matrix %d---\n", i + 1);
        printf("Give me the rows: ");
        matrix_data[i].rows = userinput(matrix_data[i].rows);
        printf("Give me the columns: ");
        matrix_data[i].columns = userinput(matrix_data[i].columns);
        matrix_data[i].numElements = TOTAL_ELEM_ARRAY;

        if (i == SECOND_MATRIX)
        {
            if ((validation_of_matMult(matrix_data) == Error))
            {
                return Error;
            }

            matrix_data[RESULT_MATRIX].rows = matrix_data[FIRST_MATRIX].rows;
            matrix_data[RESULT_MATRIX].columns = matrix_data[SECOND_MATRIX].columns;
            matrix_data[RESULT_MATRIX].numElements = TOTAL_ELEM_RESULT_MATRIX;
            matrix_data[RESULT_MATRIX].numElementsInTxt = TOTAL_ELEM_RESULT_MATRIX;

            matrix_data[RESULT_MATRIX_OMP].rows = matrix_data[FIRST_MATRIX].rows;
            matrix_data[RESULT_MATRIX_OMP].columns = matrix_data[SECOND_MATRIX].columns;
            matrix_data[RESULT_MATRIX_OMP].numElements = TOTAL_ELEM_RESULT_MATRIX;
            matrix_data[RESULT_MATRIX_OMP].numElementsInTxt = TOTAL_ELEM_RESULT_MATRIX;

        }
        
        errno_t reading = (fopen_s(&filePointer, stringArray[i], READING_MODE)); //Opening file in read mode
        filePointer_aux = filePointer; // save the file pointer read

        //Validation of the file reading process
        if (((reading != Success) || (filePointer == NULL || filePointer == 0)) && NUM_FILES > MAX_FILES)
        {
            printf("Error! Failed to open file!");
            return Error;
        }
        else
        {
            //counts the elements in the two files and saves it in matrix_data.numElementsInTxt for each file. matrix_data.numElementsInTxt contains the num of elements available in the first file
            numElementsTxt(filePointer, i, matrix_data);

            //Verify if the file contains the required num of elements to create the matrix of the size that the user requires
            if (verifySize(matrix_data, i, stringArray) != Success)
            {
                return Error;
            }

            //Proceed to create the dynamic arrays
            convertToDynamic(filePointer, i, matrix_data);

            PRINT_PARAMS(File % s opened and %lld data loaded into dynamic matrix, stringArray[i], TOTAL_ELEM_ARRAY);

            if (fclose(filePointer) != Success || filePointer == NULL || filePointer != filePointer_aux)  /* close the file prior to exiting the routine */
            {
                printf("problemas al cerrar el archivo");
                return Error;
            }

        }


    }
    return Success;
}

static void numElementsTxt(FILE* filePointer, uint8_t i, matrix_t matrix_data[])
{
    double fp;
    //PRINT(Contenido del archivo : );   //debug message
    while (feof(filePointer) == 0)
    {
        if (fscanf_s(filePointer, "%lf", &fp) == Error)
        {
            //PRINT_PARAMS(%lf,fp);         //debug message
            matrix_data[i].numElementsInTxt++;                //counting num of elements that the matrix will can have to allocate that amount of  memory
        }

    }
}


processStatus_t verifySize(matrix_t matrix_data[], uint8_t i, const char* stringArray[NUM_FILES])
{
    if (TOTAL_ELEM_ARRAY > matrix_data[i].numElementsInTxt)
    {
        printf("\n\n ------------------- Error --------------------------------");
        printf("\n\nElements of the text file %s available ----> %lld \n\n", stringArray[i], matrix_data[i].numElementsInTxt);

        printf("Elements asked ----> %lld\n\n", TOTAL_ELEM_ARRAY);
        printf("Error: Elements in the text file are not enough");
        return Error;
    }
    return Success;
}


processStatus_t convertToDynamic(FILE* filePointer, uint8_t i, matrix_t matrix_data[])
{
    /************************ DEBUG MESSAGES ***************************/
                                                                       //
    DEBUG(long long x = 0;)                                            //
        PRINT_PARAMS(array1 tam : % lld, TOTAL_ELEM_ARRAY);                //
    DEBUG(x = (long long)sizeof(double) * TOTAL_ELEM_ARRAY;)           //
        PRINT_PARAMS(tama�o en bytes : % lld, x);                          //
    //
/*******************************************************************/


    switch (i)
    {
        //Fill the first matrix
    case FIRST_MATRIX:
        //dynamic allocation
        array1 = ALIGNED_MALLOC(TOTAL_ELEM_ARRAY, ALIGNMENT_8, double); //      returns a pointer to a block of (TOTAL_ELEM_ARRAY * sizeof(*double)) memory alligned to 8 bytes
        if (array1 != NULL)
        {
            matrix_data[i].ptrArray = array1;
            readElements(filePointer, i, matrix_data);
        }
        else
        {
            printf("Memory not allocated, error ");
            perror("Malloc");

            return Error;
        }
        break;
        //Fill the second matrix
    case SECOND_MATRIX:
        array2 = ALIGNED_MALLOC(TOTAL_ELEM_ARRAY, ALIGNMENT_8, double); //      returns an 8 BYTE aligned block of memory of (total elements * 8 bytes)
        //hipMallocManaged(&array2, TOTAL_ELEM_ARRAY * sizeof(double));
        array3 = ALIGNED_MALLOC(matrix_data[RESULT_MATRIX].rows*matrix_data[RESULT_MATRIX].columns, ALIGNMENT_8, double); //      returns an 8 BYTE aligned block of memory of (total elements * 8 bytes)
        //hipMallocManaged(&array3, matrix_data[RESULT_MATRIX].rows * matrix_data[RESULT_MATRIX].columns * sizeof(double));
        array4 = ALIGNED_MALLOC(matrix_data[RESULT_MATRIX].rows * matrix_data[RESULT_MATRIX].columns, ALIGNMENT_8, double);
        DEBUG(printf("Memoria almacenada correctamente: %lld bytes\n", (long long)_aligned_msize(array3, ALIGNMENT_8, 0));)
        if ((array2 != NULL) || (array3 != NULL) || array4 != NULL)
        {
            matrix_data[i].ptrArray = array2;
            matrix_data[RESULT_MATRIX].ptrArray = array3;
            matrix_data[RESULT_MATRIX_OMP].ptrArray = array4;

            //initializing with 0's the third matrix that will allocate the result
            for (long long i = 0; i < matrix_data[RESULT_MATRIX].rows; i++)
            {
                for (long long j = 0; j < matrix_data[RESULT_MATRIX].columns; j++)
                {
                    MAT_AND_COORD(RESULT_MATRIX, i, j) = 0;
                }
            }

            for (long long i = 0; i < matrix_data[RESULT_MATRIX_OMP].rows; i++)
            {
                for (long long j = 0; j < matrix_data[RESULT_MATRIX_OMP].columns; j++)
                {
                    MAT_AND_COORD(RESULT_MATRIX_OMP, i, j) = 0;
                }
            }

            readElements(filePointer, i, matrix_data);

            DEBUG(printf("Memoria almacenada correctamente: %lld bytes\n", (long long)_aligned_msize(array1, ALIGNMENT_8, 0));)
        }
        else
        {
            printf("Memory not allocated, error ");
            perror("Malloc");

            return Error;
        }
        break;

    default:
        printf("cannot be more than 2 matrix");
        return Error;
        break;
    }

    return Success;
}





static void readElements(FILE* filePointer, uint8_t i, matrix_t matrix_data[])
{
    double fp;
    long long max_elem_allowed = TOTAL_ELEM_ARRAY;

    long long r = 0, c = 0;

    rewind(filePointer);
    while ((feof(filePointer) == 0) && max_elem_allowed != 0)
    {
        if (fscanf_s(filePointer, "%lf", &fp) == Error)
        {

            if (r < matrix_data[i].rows)
            {
                if (c < matrix_data[i].columns)
                {
                    MAT_AND_COORD(i, r, c) = fp;
                    //(MATRIX(i)[POSITION(r, c)]) = fp;
                    //(matrix_data[i].ptrArray[(r * matrix_data[i].columns) + c]) = fp;
                    c++;
                    if (c == matrix_data[i].columns)
                    {
                        c = 0;
                        r++;
                    }

                }
            }
        }
        max_elem_allowed--;
    }

    

}





processStatus_t writeElements(matrix_t matrix_data[], uint8_t which_matrix)
{
    FILE* filePointer_result; /* declare the file pointer */
    errno_t reading = Error;
    switch (which_matrix)
    {
        case SERIAL:
            reading = (processStatus_t)(fopen_s(&filePointer_result, "matrizC.txt", W_R_MODE)); //Opening file in write mode
            break;

        case CUDA:
            reading = (processStatus_t)(fopen_s(&filePointer_result, "matrizC_CUDA.txt", W_R_MODE)); //Opening file in write mode
            break;
        
        case OMP:
            reading = (processStatus_t)(fopen_s(&filePointer_result, "matrizC_OMP.txt", W_R_MODE)); //Opening file in write mode
            break;
        
        default:
            reading = Error;
            break;
    }

    
    //Validation of the file reading process
    if (((reading != Success) || (filePointer_result == NULL || filePointer_result == 0)))
    {
        printf("Error! Failed to create and open file!");
        return Error;
    }
    
    for (long long i = 0; i < matrix_data[RESULT_MATRIX].rows; i++)
    {
        for (long long j = 0; j < matrix_data[RESULT_MATRIX].columns; j++)
        {
            fprintf(filePointer_result, "%.10lf\n", MAT_AND_COORD(RESULT_MATRIX, i, j));
        }
    }

    if (fclose(filePointer_result) != Success || filePointer_result == NULL)  /* close the file prior to exiting the routine */
    {
        printf("problemas al cerrar el archivo");
        return Error;
    }
}

void compareFiles(FILE* serialFile, FILE* cudaFile, FILE* ompFile)
{
    // catching characters of the files
    char ch1 = getc(serialFile);
    char ch2 = getc(cudaFile);
    char ch3 = getc(ompFile);

    int error = 0, pos = 0, line = 1;

    // iterate loop till End Of File
    while (ch1 != EOF && ch2 != EOF && ch3 != EOF)
    {
        pos++;

        // if both variable encounters new
        // line then line variable is incremented
        // and pos variable is set to 0
        if (ch1 == '\n' && ch2 == '\n' && ch3 == '\n')
        {
            line++;
            pos = 0;
        }

        // if fetched data is not equal then
        // error is incremented
        if ((ch1 != ch2) || (ch1 != ch3))
        {
            error++;
        }

        // fetching character until end of file
        ch1 = getc(serialFile);
        ch2 = getc(cudaFile);
        ch3 = getc(ompFile);
    }

    if (error != 0)
    {
        printf("\n\n NOT PASS: Results are not equal \n\n");
    }
    else
    {
        printf("\n\n PASS: Results are equal \n\n");
    }

}