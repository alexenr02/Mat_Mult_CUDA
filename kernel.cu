#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <Windows.h>
#include "hip/hip_runtime.h"
#include ""
#include <time.h>
#include <hip/hip_fp16.h>
#include "main.cuh"
#include "file_handler.cuh"
#include "math_operation.cuh"
#include "user_input_handler.cuh"



__global__ void kernel(matrix_t matrix_data[], double* array1, double* array2, double* array3)
{
	
	long long row = (blockIdx.y * blockDim.y) + threadIdx.y;
	long long column = (blockIdx.x * blockDim.x) + threadIdx.x;
	double  temp = 0.0;

	if ((row< matrix_data[FIRST_MATRIX].rows) && (column < matrix_data[SECOND_MATRIX].columns))
	{
		for (int i = 0; i < matrix_data[FIRST_MATRIX].columns; i++)
		{
			temp += array1[(row * matrix_data[FIRST_MATRIX].columns) + i] * array2[(i * matrix_data[SECOND_MATRIX].columns) + column];
		}
	}
	array3[(row * matrix_data[RESULT_MATRIX].columns) + column] = temp;
}

int main(void)
{
	int serial_counter, CUDA_counter, OMP_counter = 0;
	double time_serial[5], time_CUDA[5], time_OMP[5];

	hipError_t cudaStatus;
	//matrix initialization
	matrix_t matrix_data[3] = { 0,0,0,0, NULL };

	if (loadTxts(matrix_data) != Success)
	{
		return Error;
	}


	/*------------------------------Serial algorithm time------------------------------------*/
																							//
	clock_t start_t, end_t, total_t;														//
	start_t = clock();																		//
	matrix_mult_serial(matrix_data);														//
	end_t = clock();																		//
	total_t = end_t - start_t;																//
	printf("\nSerial Algorithm: %f ms\n", ((((float)total_t)*1000) / CLOCKS_PER_SEC));	    //
	writeElements(matrix_data, SERIAL);														//
																							//
	/*--------------------------------------------------------------------------------------*/

	//arrays of the host
	double* h_a1 = matrix_data[FIRST_MATRIX].ptrArray;
	double* h_a2 = matrix_data[SECOND_MATRIX].ptrArray;
	double* h_a3 = matrix_data[RESULT_MATRIX].ptrArray;
	//arrays of the device
	matrix_t* d_matrix_data;
	double* d_a1;
	double* d_a2;
	double* d_a3;

	//------------------------------ Allocate memory on the device of the three arrays and validations

	cudaStatus = hipMalloc((void**)&d_matrix_data, TOTAL_MATRIX * sizeof(matrix_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc 1st array failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&d_a1, matrix_data[FIRST_MATRIX].numElements * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc 1st array failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&d_a2, matrix_data[SECOND_MATRIX].numElements * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc 2nd array failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&d_a3, matrix_data[RESULT_MATRIX].numElements * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc result array failed!");
		goto Error;
	}

	//------------------------------ copy memory from host to Device and validations

	cudaStatus = hipMemcpy(d_matrix_data, matrix_data, TOTAL_MATRIX * sizeof(matrix_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy matrix data failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(d_a1, h_a1, matrix_data[FIRST_MATRIX].numElements * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 1st array failed!");
		goto Error;
	}
	//copy memory from host to Device
	cudaStatus = hipMemcpy(d_a2, h_a2, matrix_data[SECOND_MATRIX].numElements * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 2nd array failed!");
		goto Error;
	}
	//copy memory from host to Device
	cudaStatus = hipMemcpy(d_a3, h_a3, matrix_data[RESULT_MATRIX].numElements * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy result array failed!");
		goto Error;
	}

	// Metrics
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//hipDeviceProp_t prop;
	////Declare variables
	//hipGetDeviceProperties(&prop, 0);
	//int maxBLOCK_SIZE_X = prop.maxThreadsDim[0];
	//int maxBLOCK_SIZE_Y = prop.maxThreadsDim[1];
	//int maxGRID_SIZE_X = prop.maxGridSize[0];
	//int maxGRID_SIZE_Y = prop.maxGridSize[1];
	//int maxThreadsAvailable_x = maxBLOCK_SIZE_X * maxGRID_SIZE_X;
	//int maxThreadsAvailable_y = maxBLOCK_SIZE_Y * maxGRID_SIZE_Y;

	int blockdim = WARP; //Num of threads = WARP size
	dim3 threadsPerBlock(blockdim, blockdim);
	dim3 blocksPerGrid(1,1);

	blocksPerGrid.x = ceil(double(matrix_data[SECOND_MATRIX].columns)/threadsPerBlock.x);
	blocksPerGrid.y = ceil(double(matrix_data[FIRST_MATRIX].rows)/threadsPerBlock.y);
	
	/*------------------------------CUDA algorithm time------------------------------------*/
																							//
	hipEventRecord(start);																	//
	//Launch the kernel																		//
	kernel << <  blocksPerGrid, threadsPerBlock >> > (d_matrix_data, d_a1, d_a2, d_a3);		//
	hipEventRecord(stop);																	//
	writeElements(matrix_data, CUDA);														//
																							//
	/*--------------------------------------------------------------------------------------*/

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	//hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);
		goto Error;
	}
	hipEventSynchronize(stop);

	// Copy data back to host
	cudaStatus = hipMemcpy(h_a3, d_a3, matrix_data[RESULT_MATRIX].numElements * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	matrix_data[RESULT_MATRIX].ptrArray = h_a3;

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("\n(Total CUDA time: %lf ms)\n", milliseconds);
	

Error:
	ending_program
	return 0;
}
